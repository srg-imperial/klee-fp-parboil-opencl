#include "hip/hip_runtime.h"
/***************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/
/*
 * CUDA accelerated coulombic potential grid test code
 *   John E. Stone <johns@ks.uiuc.edu>
 *   http://www.ks.uiuc.edu/~johns/
 *
 * Coulombic potential grid calculation microbenchmark based on the time
 * consuming portions of the 'cionize' ion placement tool.
 */

#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#include <common.h>

#include "cuenergy.h"

/* initatoms()
 * Store a pseudorandom arrangement of point charges in *atombuf.
 */
static int
initatoms(float **atombuf, int count, dim3 volsize, float gridspacing) {
  dim3 size;
  int i;
  float *atoms;

  srand(54321);			// Ensure that atom placement is repeatable

  atoms = (float *) malloc(count * 4 * sizeof(float));
  *atombuf = atoms;

  // compute grid dimensions in angstroms
  size.x = gridspacing * volsize.x;
  size.y = gridspacing * volsize.y;
  size.z = gridspacing * volsize.z;

  for (i=0; i<count; i++) {
    int addr = i * 4;
    atoms[addr    ] = (rand() / (float) RAND_MAX) * size.x; 
    atoms[addr + 1] = (rand() / (float) RAND_MAX) * size.y; 
    atoms[addr + 2] = (rand() / (float) RAND_MAX) * size.z; 
    atoms[addr + 3] = ((rand() / (float) RAND_MAX) * 2.0) - 1.0;  // charge
  }  

  return 0;
}

/* writeenergy()
 * Write part of the energy array to an output file for verification.
 */
static int
writeenergy(char *filename, float *energy, dim3 volsize)
{
  FILE *outfile;
  int x, y;

  outfile = fopen(filename, "w");
  if (outfile == NULL) {
    fputs("Cannot open output file\n", stderr);
    return -1;
    }

  /* Print the execution parameters */
  fprintf(outfile, "%d %d %d %d\n", volsize.x, volsize.y, volsize.z, ATOMCOUNT);

  /* Print a checksum */
  {
    double sum = 0.0;

    for (y = 0; y < volsize.y; y++) {
      for (x = 0; x < volsize.x; x++) {
        double t = energy[y*volsize.x+x];
        t = fmax(-20.0, fmin(20.0, t));
    	sum += t;
      }
    }
    fprintf(outfile, "%.4g\n", sum);
  }
  
  /* Print several rows of the computed data */
  for (y = 0; y < 17; y++) {
    for (x = 0; x < volsize.x; x++) {
      int addr = y * volsize.x + x;
      fprintf(outfile, "%.4g ", energy[addr]);
    }
    fprintf(outfile, "\n");
  }

  fclose(outfile);

  return 0;
}

int main(int argc, char** argv) {

  float *energy = NULL;		// Output of device calculation
  float *atoms = NULL;
  dim3 volsize, Gsz, Bsz;

  // number of atoms to simulate
  int atomcount = ATOMCOUNT;

  // voxel spacing
  const float gridspacing = 0.1;

  // Size of buffer on GPU
  int volmemsz;

  //printf("CUDA accelerated coulombic potential microbenchmark\n");
  //printf("Original version by John E. Stone <johns@ks.uiuc.edu>\n");
  //printf("This version maintained by Chris Rodrigues\n");

  // setup energy grid size
  volsize.x = VOLSIZEX;
  volsize.y = VOLSIZEY;
  volsize.z = 1;

  // use user-specified values if provided
  getCmdLineParamInt("-volx", argc, argv, (int*)&volsize.x);
  getCmdLineParamInt("-voly", argc, argv, (int*)&volsize.y);
  getCmdLineParamInt("-atoms", argc, argv, &atomcount);

  // setup CUDA grid and block sizes
  Bsz.x = BLOCKSIZEX;		// each thread does multiple Xs
  Bsz.y = BLOCKSIZEY;
  Bsz.z = 1;
  // use user-specified block sizes if provided
  getCmdLineParamInt("-localx", argc, argv, (int*)&Bsz.x);
  getCmdLineParamInt("-localy", argc, argv, (int*)&Bsz.y);

  Gsz.x = volsize.x / (Bsz.x * UNROLLX); // each thread does multiple Xs
  Gsz.y = volsize.y / Bsz.y; 
  Gsz.z = volsize.z / Bsz.z; 

#if 0
  printf("Grid size: %d x %d x %d\n", volsize.x, volsize.y, volsize.z);
  printf("Running kernel(atoms:%d, gridspacing %g, z %d)\n", atomcount, gridspacing, 0);
#endif

  // allocate and initialize atom coordinates and charges
  if (initatoms(&atoms, atomcount, volsize, gridspacing))
    return -1;

  // allocate and initialize the GPU output array
  volmemsz = sizeof(float) * volsize.x * volsize.y * volsize.z;

#ifdef PROFILING
  // profile kernel executions
  int numIterations = (atomcount%MAXATOMS) ? (atomcount/MAXATOMS + 1) : (atomcount/MAXATOMS);
  hipEvent_t *evStart = (hipEvent_t*) malloc (sizeof(hipEvent_t) * numIterations);
  hipEvent_t *evStop  = (hipEvent_t*) malloc (sizeof(hipEvent_t) * numIterations);
  for (int i=0; i<numIterations; i++)
  {
    hipEventCreate(&evStart[i]);
    hipEventCreate(&evStop[i]);
  }

  inf_timer tMain;
  startTimer(&tMain);
#endif

  // Main computation
  {
    float *d_output = NULL;	// Output on device
    int iterations=0;
    int atomstart;

    hipMalloc((void**)&d_output, volmemsz);
    hipMemset(d_output, 0, volmemsz);

    for (atomstart=0; atomstart<atomcount; atomstart+=MAXATOMS) {   
      int atomsremaining = atomcount - atomstart;
      int runatoms = (atomsremaining > MAXATOMS) ? MAXATOMS : atomsremaining;
      iterations++;

      // copy the atoms to the GPU
      if (copyatomstoconstbuf(atoms + 4*atomstart, runatoms, 0*gridspacing)) 
	return -1;

      //if (parameters->synchronizeGpu) hipDeviceSynchronize();

#ifdef PROFILING
      hipEventRecord(evStart[iterations-1], 0);
#endif
 
      // RUN the kernel...
      cenergy<<<Gsz, Bsz, 0>>>(runatoms, 0.1, d_output);

#ifdef PROFILING
      hipEventRecord(evStop[iterations-1], 0);
#endif

    }

    // Copy the GPU output data back to the host and use/store it..
    energy = (float *) malloc(volmemsz);
    hipMemcpy(energy, d_output, volmemsz,  hipMemcpyDeviceToHost);

    hipFree(d_output);
  }

#ifdef PROFILING
  hipDeviceSynchronize();
  stopTimer(&tMain);
  printf ("main computation: %10fms\n", elapsedTime(tMain));
#endif

  /* Print a subset of the results to a file */
  /*if (parameters->outFile) {
    pb_SwitchToTimer(&timers, pb_TimerID_IO);
    if (writeenergy(parameters->outFile, energy, volsize) == -1)
      return -1;
    pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);
  }*/

  free(atoms);
  free(energy);

#ifdef PROFILING
  // output kernel runtimes
  float avg;
  for (int i=0; i<numIterations; i++)
  {
    float diff;
    hipEventElapsedTime(&diff, evStart[i], evStop[i]);
    printf ("kernel %d: %fms\n", i, diff);

    avg += diff;
  }
  printf ("kernel avg: %fms\n", avg/numIterations);

  free(evStart);
  free(evStop);
#endif

  return 0;
}



