/***************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

/*
 * CUDA code for creating the FHD data structure for fast convolution-based 
 * Hessian multiplication for arbitrary k-space trajectories.
 * 
 * recommended g++ options:
 *   -O3 -lm -ffast-math -funroll-all-loops
 *
 * Inputs:
 * kx - VECTOR of kx values, same length as ky and kz
 * ky - VECTOR of ky values, same length as kx and kz
 * kz - VECTOR of kz values, same length as kx and ky
 * x  - VECTOR of x values, same length as y and z
 * y  - VECTOR of y values, same length as x and z
 * z  - VECTOR of z values, same length as x and y
 * phi - VECTOR of the Fourier transform of the spatial basis 
 *     function, evaluated at [kx, ky, kz].  Same length as kx, ky, and kz.
 */

#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <sys/time.h>

#include <common.h>

#include "file.h"
#include "computeFH.cu"

static void
setupMemoryGPU(int num, int size, float*& dev_ptr, float*& host_ptr)
{
  hipMalloc ((void **) &dev_ptr, num * size);
  CUDA_ERRCK;
  hipMemcpy (dev_ptr, host_ptr, num * size, hipMemcpyHostToDevice);
  CUDA_ERRCK;
}

static void
cleanupMemoryGPU(int num, int size, float *& dev_ptr, float * host_ptr)
{
  hipMemcpy (host_ptr, dev_ptr, num * size, hipMemcpyDeviceToHost);
  CUDA_ERRCK;
  hipFree(dev_ptr);
  CUDA_ERRCK;
}

int
main (int argc, char *argv[])
{
  int numX, numK;		/* Number of X and K values */
  int original_numK, original_numX;		/* Number of K and X values in input file */
  float *kx, *ky, *kz;		/* K trajectory (3D vectors) */
  float *x, *y, *z;		/* X coordinates (3D vectors) */
  float *phiR, *phiI;		/* Phi values (complex) */
  float *dR, *dI;		/* D values (complex) */
  float *realRhoPhi, *imagRhoPhi;  /* RhoPhi values (complex) */
  float *outI, *outR;		/* Output signal (complex) */
  kValues* kVals;		/* Copy of X and RhoPhi.  Its
				 * data layout has better cache
				 * performance. */

  /* Read in data */
  //inputData(params->inpFiles[0],
  inputData(argv[1],
	    &original_numK, &original_numX,
	    &kx, &ky, &kz,
	    &x, &y, &z,
	    &phiR, &phiI,
	    &dR, &dI);

  /* Reduce the number of k-space samples if a number is given
   * on the command line */
  if (getCmdLineParamInt("-numK", argc, argv, &numK))
    numK = MIN(numK, original_numK);
  else
    numK = original_numK;

  if (getCmdLineParamInt("-numX", argc, argv, &numX))
    numX = MIN(numX, original_numX);
  else
    numX = original_numX;

  printf("%d pixels in output; %d samples in trajectory; using %d samples\n",
         numX, original_numK, numK);

  /* Create CPU data structures */
  createDataStructs(numK, numX, realRhoPhi, imagRhoPhi, outR, outI);
  kVals = (kValues*)calloc(numK, sizeof (kValues));

  inf_timer ifGPU;
  startTimer(&ifGPU);

  /* GPU section 1 (precompute Rho, Phi)*/
  {
    /* Mirror several data structures on the device */
    float *phiR_d, *phiI_d;
    float *dR_d, *dI_d;
    float *realRhoPhi_d, *imagRhoPhi_d;

    setupMemoryGPU(numK, sizeof(float), phiR_d, phiR);
    setupMemoryGPU(numK, sizeof(float), phiI_d, phiI);
    setupMemoryGPU(numK, sizeof(float), dR_d, dR);
    setupMemoryGPU(numK, sizeof(float), dI_d, dI);
    hipMalloc((void **)&realRhoPhi_d, numK * sizeof(float));
    CUDA_ERRCK;
    hipMalloc((void **)&imagRhoPhi_d, numK * sizeof(float));
    CUDA_ERRCK;

    /* Pre-compute the values of rhoPhi on the GPU */
    computeRhoPhi_GPU(argc, argv, numK, phiR_d, phiI_d, dR_d, dI_d, 
		      realRhoPhi_d, imagRhoPhi_d);

    cleanupMemoryGPU(numK, sizeof(float), realRhoPhi_d, realRhoPhi);
    cleanupMemoryGPU(numK, sizeof(float), imagRhoPhi_d, imagRhoPhi);
    hipFree(phiR_d);
    hipFree(phiI_d);
    hipFree(dR_d);
    hipFree(dI_d);
  }

  hipDeviceSynchronize();
  stopTimer(&ifGPU);
  printf ("loop1 1: %fms\n", elapsedTime(ifGPU));

  /* Fill in kVals values */
  for (int k = 0; k < numK; k++) {
    kVals[k].Kx = kx[k];
    kVals[k].Ky = ky[k];
    kVals[k].Kz = kz[k];
    kVals[k].RhoPhiR = realRhoPhi[k];
    kVals[k].RhoPhiI = imagRhoPhi[k];
  }

  startTimer(&ifGPU);

  /* GPU section 2 (compute FH)*/
  {
    float *x_d, *y_d, *z_d;
    float *outI_d, *outR_d;

    /* Mirror several data structures on the device */
    setupMemoryGPU(numX, sizeof(float), x_d, x);
    setupMemoryGPU(numX, sizeof(float), y_d, y);
    setupMemoryGPU(numX, sizeof(float), z_d, z);

    // Zero out initial values of outR and outI.
    // GPU veiws these arrays as initialized (cleared) accumulators.
    hipMalloc((void **)&outR_d, numX * sizeof(float));
    CUDA_ERRCK;
    hipMemset(outR_d, 0, numX * sizeof(float));
    CUDA_ERRCK;
    hipMalloc((void **)&outI_d, numX * sizeof(float));
    CUDA_ERRCK;
    hipMemset(outI_d, 0, numX * sizeof(float));
    CUDA_ERRCK;

    /* Compute FH on the GPU (main computation) */
    computeFH_GPU(argc, argv, numK, numX, x_d, y_d, z_d, kVals, outR_d, outI_d);

    /* Release memory on GPU */
    cleanupMemoryGPU(numX, sizeof(float), outR_d, outR);
    cleanupMemoryGPU(numX, sizeof(float), outI_d, outI);

    hipFree(x_d);
    hipFree(y_d);
    hipFree(z_d);
  }

  stopTimer(&ifGPU);
  printf ("loop 2: %fms\n", elapsedTime(ifGPU));

  /* Write result to file */
  //outputData(params->outFile, outR, outI, numX);

  free (kx);
  free (ky);
  free (kz);
  free (x);
  free (y);
  free (z);
  free (phiR);
  free (phiI);
  free (dR);
  free (dI);
  free (realRhoPhi);
  free (imagRhoPhi);
  free (kVals);
  free (outR);
  free (outI);

#ifdef PROFILING
  // output kernel runtimes
  float avg;
  float diff;

  printf ("\n----------------------------\n");

  hipEventElapsedTime(&diff, evRhoPhiStart, evRhoPhiStop);
  printf ("RhoPhi: %fms\n", diff);

  avg=0.0;
  for (int i=0; i<numFH; i++)
  {
    hipEventElapsedTime(&diff, evFHStart[i], evFHStop[i]);
    printf ("FH %d: %fms\n", i, diff);

    avg += diff;
  }
  printf ("FH avg: %fms\n", avg/numFH);

  free(evFHStart);
  free(evFHStop);
#endif

  return 0;
}
